#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <assert.h>

// cuda
#include <nvToolsExt.h>

//------------------------------------------------------------------------------
namespace internal {
void task()
{
    nvtxRangePush( __func__ );
    int r = rand() % 1000;
    usleep( r );
    nvtxRangePop();
}
}

//------------------------------------------------------------------------------
void task()
{
    nvtxRangePush( __func__ );
    int r = rand() % 1000;
    usleep( r );
    nvtxRangePop();
}

//------------------------------------------------------------------------------
int main()
{
    printf( "hello\n" );
    task();
    
    int size = 1024;
    double *src, *dst;
    hipError_t err;
    err = hipMalloc( &src, size );
    assert( err == hipSuccess );
    err = hipMalloc( &dst, size );
    assert( err == hipSuccess );
    err = hipMemcpy( dst, src, size, hipMemcpyDefault );
    assert( err == hipSuccess );
    err = hipFree( src );
    assert( err == hipSuccess );
    err = hipFree( dst );
    assert( err == hipSuccess );
    
    internal::task();
    return 0;
}
